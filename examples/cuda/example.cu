/** @file example.cpp
 *  @brief Usage example for the C++ API
 */

#include "sphericart_cuda.hpp"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;
using namespace sphericart::cuda;

/*host macro that checks for errors in CUDA calls, and prints the file + line
 * and error string if one occurs
 */
#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t cudaStatus = (call);                                       \
        if (cudaStatus != hipSuccess) {                                       \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__       \
                      << " - " << hipGetErrorString(cudaStatus) << std::endl; \
            hipDeviceReset();                                                 \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

int main() {
    /* ===== set up the calculation ===== */

    // hard-coded parameters for the example
    size_t n_samples = 10000;
    size_t l_max = 10;

    // initializes samples
    auto xyz = std::vector<double>(n_samples * 3, 0.0);
    for (size_t i = 0; i < n_samples * 3; ++i) {
        xyz[i] = (double)rand() / (double)RAND_MAX * 2.0 - 1.0;
    }

    // to avoid unnecessary allocations, calculators can use pre-allocated
    // memory, one also can provide uninitialized vectors that will be
    // automatically reshaped
    auto sph = std::vector<double>(n_samples * (l_max + 1) * (l_max + 1), 0.0);
    auto dsph =
        std::vector<double>(n_samples * 3 * (l_max + 1) * (l_max + 1), 0.0);
    auto ddsph =
        std::vector<double>(n_samples * 3 * 3 * (l_max + 1) * (l_max + 1), 0.0);

    /* ===== API calls ===== */

    // internal buffers and numerical factors are initalized at construction
    sphericart::cuda::SphericalHarmonics<double> calculator_cuda(l_max);

    double *xyz_cuda;
    CUDA_CHECK(hipMalloc(&xyz_cuda, n_samples * 3 * sizeof(double)));
    CUDA_CHECK(hipMemcpy(xyz_cuda, xyz.data(), n_samples * 3 * sizeof(double),
                          hipMemcpyHostToDevice));
    double *sph_cuda;
    CUDA_CHECK(hipMalloc(&sph_cuda, n_samples * (l_max + 1) * (l_max + 1) *
                                         sizeof(double)));

    calculator_cuda.compute(xyz_cuda, n_samples, false, false,
                            sph_cuda); // no gradients */

    CUDA_CHECK(
        hipMemcpy(sph.data(), sph_cuda,
                   n_samples * (l_max + 1) * (l_max + 1) * sizeof(double),
                   hipMemcpyDeviceToHost));

    for (int i = 0; i < 4; i++) {
        std::cout << sph[i] << std::endl;
    }

    return 0;
}
