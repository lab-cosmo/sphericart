/** @file example.cpp
 *  @brief Usage example for the C++ API
 */
#include "sphericart.hpp"
#include "sphericart_cuda.hpp"
#include <chrono>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;
using namespace sphericart;
using namespace sphericart::cuda;

/*host macro that checks for errors in CUDA calls, and prints the file + line
 * and error string if one occurs
 */
#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t cudaStatus = (call);                                       \
        if (cudaStatus != hipSuccess) {                                       \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__       \
                      << " - " << hipGetErrorString(cudaStatus) << std::endl; \
            hipDeviceReset();                                                 \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

template <class scalar_t> void timing() {
    /* ===== set up the calculation ===== */

    // hard-coded parameters for the example
    size_t n_samples = 100000;
    size_t l_max = 32;

    // initializes samples
    auto xyz = std::vector<scalar_t>(n_samples * 3, 0.0);
    for (size_t i = 0; i < n_samples * 3; ++i) {
        xyz[i] = (scalar_t)rand() / (scalar_t)RAND_MAX * 2.0 - 1.0;
    }

    // to avoid unnecessary allocations, calculators can use pre-allocated
    // memory, one also can provide uninitialized vectors that will be
    // automatically reshaped
    auto sph =
        std::vector<scalar_t>(n_samples * (l_max + 1) * (l_max + 1), 0.0);

    auto sph_cpu =
        std::vector<scalar_t>(n_samples * (l_max + 1) * (l_max + 1), 0.0);
    auto dsph =
        std::vector<scalar_t>(n_samples * 3 * (l_max + 1) * (l_max + 1), 0.0);
    auto ddsph = std::vector<scalar_t>(
        n_samples * 3 * 3 * (l_max + 1) * (l_max + 1), 0.0);

    /* ===== API calls ===== */

    // internal buffers and numerical factors are initalized at construction
    sphericart::cuda::SphericalHarmonics<scalar_t> calculator_cuda(l_max);

    scalar_t *xyz_cuda;
    CUDA_CHECK(hipMalloc(&xyz_cuda, n_samples * 3 * sizeof(scalar_t)));
    CUDA_CHECK(hipMemcpy(xyz_cuda, xyz.data(),
                          n_samples * 3 * sizeof(scalar_t),
                          hipMemcpyHostToDevice));
    scalar_t *sph_cuda;
    CUDA_CHECK(hipMalloc(&sph_cuda, n_samples * (l_max + 1) * (l_max + 1) *
                                         sizeof(scalar_t)));

    scalar_t *dsph_cuda;
    CUDA_CHECK(hipMalloc(&dsph_cuda, 3 * n_samples * (l_max + 1) *
                                          (l_max + 1) * sizeof(scalar_t)));
    for (int i = 0; i < 5; i++) {
        calculator_cuda.compute_with_gradients(xyz_cuda, n_samples, sph_cuda,
                                               dsph_cuda);
        // calculator_cuda.compute(xyz_cuda, n_samples, sph_cuda); // no
        //  gradients
    }

    std::cout << "-----------------" << std::endl;

    auto start = std::chrono::high_resolution_clock::now();

    calculator_cuda.compute_with_gradients(xyz_cuda, n_samples, sph_cuda,
                                           dsph_cuda);

    // calculator_cuda.compute(xyz_cuda, n_samples, sph_cuda); // no gradients

    // Record the end time
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the duration
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

    // Print the duration in microseconds
    std::cout << "Time taken by function: " << duration.count()
              << " nanoseconds" << std::endl;
    std::cout << "" << ((double)duration.count()) / ((double)n_samples)
              << " ns/sample" << std::endl;
    // */
    CUDA_CHECK(
        hipMemcpy(sph.data(), sph_cuda,
                   n_samples * (l_max + 1) * (l_max + 1) * sizeof(scalar_t),
                   hipMemcpyDeviceToHost));

    auto calculator = sphericart::SphericalHarmonics<scalar_t>(l_max);

    calculator.compute(xyz, sph_cpu);

    /*for (int i = 0; i < n_samples; i++) {
        std::cout << "sample: " << xyz[i * 3 + 0] << " " << xyz[i * 3 + 1]
                  << " " << xyz[i * 3 + 2] << std::endl;
    }

    for (int i = 0; i < n_samples * (l_max + 1) * (l_max + 1); i++) {
        std::cout << sph[i] << " " << sph_cpu[i] << std::endl;
    }*/
}

int main() {
    timing<double>();

    return 0;
}
