#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define _SPHERICART_INTERNAL_IMPLEMENTATION
#define CUDA_DEVICE_PREFIX __device__

#include "cuda_base.hpp"

#define HARDCODED_LMAX 1

/* MASK used for warp reductions */
#define FULL_MASK 0xffffffff

#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t cudaStatus = (call);                                       \
        if (cudaStatus != hipSuccess) {                                       \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__       \
                      << " - " << hipGetErrorString(cudaStatus) << std::endl; \
            hipDeviceReset();                                                 \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

#define CUDA_CHECK_KERNEL()                                                    \
    do {                                                                       \
        hipDeviceSynchronize();                                               \
        hipError_t err = hipGetLastError();                                  \
        if (err != hipSuccess) {                                              \
            fprintf(stderr,                                                    \
                    "CUDA error after kernel launch in %s at line %d: %s\n",   \
                    __FILE__, __LINE__, hipGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

/*
    Computes the index for buffer values which are shared across GRID_DIM_Y
*/
__device__ int get_index(int i) { return i * blockDim.y + threadIdx.y; }

/*
    Clears the shared memory buffers for the spherical harmonics and gradients
   if required.
*/
template <typename scalar_t>
__device__ inline void
clear_buffers(int nelements, scalar_t *sph, scalar_t *dsph_x, scalar_t *dsph_y,
              scalar_t *dsph_z,

              scalar_t *dsph_dxdx, scalar_t *dsph_dxdy, scalar_t *dsph_dxdz,

              scalar_t *dsph_dydx, scalar_t *dsph_dydy, scalar_t *dsph_dydz,

              scalar_t *dsph_dzdx, scalar_t *dsph_dzdy, scalar_t *dsph_dzdz,
              bool requires_grad, bool requires_hessian) {
    for (int i = threadIdx.x; i < nelements; i += blockDim.x) {
        sph[get_index(i)] = 0.0;

        if (requires_grad) {
            dsph_x[get_index(i)] = 0.0;
            dsph_y[get_index(i)] = 0.0;
            dsph_z[get_index(i)] = 0.0;
        }

        if (requires_hessian) {
            dsph_dxdx[get_index(i)] = 0.0;
            dsph_dxdy[get_index(i)] = 0.0;
            dsph_dxdz[get_index(i)] = 0.0;

            dsph_dydx[get_index(i)] = 0.0;
            dsph_dydy[get_index(i)] = 0.0;
            dsph_dydz[get_index(i)] = 0.0;

            dsph_dzdx[get_index(i)] = 0.0;
            dsph_dzdy[get_index(i)] = 0.0;
            dsph_dzdz[get_index(i)] = 0.0;
        }
    }
    __syncthreads();
}

/*
    Writes out the shared memory buffers to global memory, as well as applying
   normalisation if necessary.
*/
template <typename scalar_t>
__device__ inline void write_buffers(
    size_t edge_idx, size_t nedges, scalar_t x, scalar_t y, scalar_t z,
    scalar_t ir, int n_elements, int offset, scalar_t *buffer_sph,

    scalar_t *buffer_dsph_x, scalar_t *buffer_dsph_y, scalar_t *buffer_dsph_z,

    scalar_t *buffer_dsph_dxdx, scalar_t *buffer_dsph_dxdy,
    scalar_t *buffer_dsph_dxdz,

    scalar_t *buffer_dsph_dydx, scalar_t *buffer_dsph_dydy,
    scalar_t *buffer_dsph_dydz,

    scalar_t *buffer_dsph_dzdx, scalar_t *buffer_dsph_dzdy,
    scalar_t *buffer_dsph_dzdz, scalar_t *sph, scalar_t *dsph, scalar_t *ddsph,
    size_t n_total, bool requires_grad, bool requires_hessian, bool normalize) {
    if (edge_idx < nedges) {
        for (int i = threadIdx.x; i < n_elements; i += blockDim.x) {

            sph[edge_idx * n_total + offset + i] = buffer_sph[get_index(i)];

            if (requires_hessian) {
                auto tmp_dx = buffer_dsph_x[get_index(i)];
                auto tmp_dy = buffer_dsph_y[get_index(i)];
                auto tmp_dz = buffer_dsph_z[get_index(i)];

                auto tmp_dxdx = buffer_dsph_dxdx[get_index(i)];
                auto tmp_dxdy = buffer_dsph_dxdy[get_index(i)];
                auto tmp_dxdz = buffer_dsph_dxdz[get_index(i)];

                auto tmp_dydx = buffer_dsph_dydx[get_index(i)];
                auto tmp_dydy = buffer_dsph_dydy[get_index(i)];
                auto tmp_dydz = buffer_dsph_dydz[get_index(i)];

                auto tmp_dzdx = buffer_dsph_dzdx[get_index(i)];
                auto tmp_dzdy = buffer_dsph_dzdy[get_index(i)];
                auto tmp_dzdz = buffer_dsph_dzdz[get_index(i)];

                if (normalize) {
                    auto tmp = (tmp_dx * x + tmp_dy * y + tmp_dz * z);

                    auto tmpx = x * tmp_dxdx + y * tmp_dydx + z * tmp_dzdx;
                    auto tmpy = x * tmp_dxdy + y * tmp_dydy + z * tmp_dydz;
                    auto tmpz = x * tmp_dxdz + y * tmp_dydz + z * tmp_dzdz;
                    auto tmp2 = x * x * tmp_dxdx + y * y * tmp_dydy +
                                z * z * tmp_dzdz + 2 * x * y * tmp_dxdy +
                                2 * x * z * tmp_dxdz + 2 * y * z * tmp_dydz;

                    tmp_dxdx = (-2 * x * tmpx + tmp_dxdx + 3 * x * x * tmp -
                                tmp - 2 * x * tmp_dx + x * x * tmp2) *
                               (ir * ir);
                    tmp_dydy = (-2 * y * tmpy + tmp_dydy + 3 * y * y * tmp -
                                tmp - 2 * y * tmp_dy + y * y * tmp2) *
                               (ir * ir);
                    tmp_dzdz = (-2 * z * tmpz + tmp_dzdz + 3 * z * z * tmp -
                                tmp - 2 * z * tmp_dz + z * z * tmp2) *
                               (ir * ir);

                    tmp_dxdy = tmp_dydx =
                        (-x * tmpy - y * tmpx + tmp_dxdy + 3 * x * y * tmp -
                         x * tmp_dy - y * tmp_dx + x * y * tmp2) *
                        (ir * ir);
                    tmp_dxdz = tmp_dzdx =
                        (-x * tmpz - z * tmpx + tmp_dxdz + 3 * x * z * tmp -
                         x * tmp_dz - z * tmp_dx + x * z * tmp2) *
                        (ir * ir);
                    tmp_dzdy = tmp_dydz =
                        (-z * tmpy - y * tmpz + tmp_dzdy + 3 * y * z * tmp -
                         z * tmp_dy - y * tmp_dz + y * z * tmp2) *
                        (ir * ir);
                }

                ddsph[edge_idx * 9 * n_total + 0 * 3 * n_total + 0 * n_total +
                      offset + i] = tmp_dxdx;
                ddsph[edge_idx * 9 * n_total + 0 * 3 * n_total + 1 * n_total +
                      offset + i] = tmp_dxdy;
                ddsph[edge_idx * 9 * n_total + 0 * 3 * n_total + 2 * n_total +
                      offset + i] = tmp_dxdz;

                ddsph[edge_idx * 9 * n_total + 1 * 3 * n_total + 0 * n_total +
                      offset + i] = tmp_dydx;
                ddsph[edge_idx * 9 * n_total + 1 * 3 * n_total + 1 * n_total +
                      offset + i] = tmp_dydy;
                ddsph[edge_idx * 9 * n_total + 1 * 3 * n_total + 2 * n_total +
                      offset + i] = tmp_dydz;

                ddsph[edge_idx * 9 * n_total + 2 * 3 * n_total + 0 * n_total +
                      offset + i] = tmp_dzdx;
                ddsph[edge_idx * 9 * n_total + 2 * 3 * n_total + 1 * n_total +
                      offset + i] = tmp_dzdy;
                ddsph[edge_idx * 9 * n_total + 2 * 3 * n_total + 2 * n_total +
                      offset + i] = tmp_dzdz;
            }

            if (requires_grad) {
                auto tmp_dx = buffer_dsph_x[get_index(i)];
                auto tmp_dy = buffer_dsph_y[get_index(i)];
                auto tmp_dz = buffer_dsph_z[get_index(i)];

                // corrects derivatives for normalization
                if (normalize) {
                    auto tmp = (tmp_dx * x + tmp_dy * y + tmp_dz * z);

                    tmp_dx = (tmp_dx - x * tmp) * ir;
                    tmp_dy = (tmp_dy - y * tmp) * ir;
                    tmp_dz = (tmp_dz - z * tmp) * ir;
                }

                dsph[edge_idx * 3 * n_total + 0 * n_total + offset + i] =
                    tmp_dx;
                dsph[edge_idx * 3 * n_total + 1 * n_total + offset + i] =
                    tmp_dy;
                dsph[edge_idx * 3 * n_total + 2 * n_total + offset + i] =
                    tmp_dz;
            }
        }
    }
}

/*
    CUDA kernel for computing Cartesian spherical harmonics and their
   derivatives.
*/
template <typename scalar_t>
__global__ void spherical_harmonics_kernel(
    const scalar_t *__restrict__ xyz, int nedges,
    const scalar_t *__restrict__ prefactors, int nprefactors, int lmax,
    int ntotal, bool requires_grad, bool requires_hessian, bool normalize,
    scalar_t *__restrict__ sph, scalar_t *__restrict__ dsph,
    scalar_t *__restrict__ ddsph) {

    extern __shared__ char buffer[];

    size_t offset = 0;

    scalar_t *buffer_c = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += blockDim.y * (lmax + 1) * sizeof(scalar_t);
    scalar_t *buffer_s = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += blockDim.y * (lmax + 1) * sizeof(scalar_t);
    scalar_t *buffer_twomz = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += blockDim.y * (lmax + 1) * sizeof(scalar_t);
    scalar_t *buffer_prefactors = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += nprefactors * sizeof(scalar_t);

    int nl = max(static_cast<int>((HARDCODED_LMAX + 1) * (HARDCODED_LMAX + 1)),
                 2 * lmax + 1);

    scalar_t *buffer_sph = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += blockDim.y * nl * sizeof(scalar_t);

    scalar_t *buffer_dsph_x;
    scalar_t *buffer_dsph_y;
    scalar_t *buffer_dsph_z;

    if (requires_grad) {
        buffer_dsph_x = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_y = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_z = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
    }

    scalar_t *buffer_dsph_dxdx;
    scalar_t *buffer_dsph_dxdy;
    scalar_t *buffer_dsph_dxdz;
    scalar_t *buffer_dsph_dydx;
    scalar_t *buffer_dsph_dydy;
    scalar_t *buffer_dsph_dydz;
    scalar_t *buffer_dsph_dzdx;
    scalar_t *buffer_dsph_dzdy;
    scalar_t *buffer_dsph_dzdz;

    if (requires_hessian) {
        buffer_dsph_dxdx = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dxdy = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dxdz = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);

        buffer_dsph_dydx = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dydy = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dydz = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);

        buffer_dsph_dzdx = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dzdy = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
        buffer_dsph_dzdz = reinterpret_cast<scalar_t *>(buffer + offset);
        offset += blockDim.y * nl * sizeof(scalar_t);
    }

    size_t edge_idx = blockIdx.x * blockDim.y + threadIdx.y;

    scalar_t x = 0.0;
    scalar_t y = 0.0;
    scalar_t z = 0.0;

    scalar_t x2 = 0.0;
    scalar_t y2 = 0.0;
    scalar_t z2 = 0.0;

    if (threadIdx.y == 0) {
        for (int i = threadIdx.x; i < nprefactors; i += blockDim.x) {
            buffer_prefactors[i] = prefactors[i];
        }
    }
    __syncthreads();

    if (edge_idx < nedges) {
        x = xyz[edge_idx * 3 + 0];
        y = xyz[edge_idx * 3 + 1];
        z = xyz[edge_idx * 3 + 2];

        x2 = x * x;
        y2 = y * y;
        z2 = z * z;
    }

    scalar_t ir = 0.0;

    if (normalize) {
        if (edge_idx < nedges) {
            auto ir2 = 1.0 / (x2 + y2 + z2);
            ir = sqrt(ir2);
            x *= ir;
            y *= ir;
            z *= ir;
            x2 *= ir2;
            y2 *= ir2;
            z2 *= ir2;
        }
    }

    auto rxy = x2 + y2;
    auto twoz = 2 * z;
    if (threadIdx.x == 0) {
        buffer_c[get_index(0)] = 1.0;
        buffer_s[get_index(0)] = 0.0;
        buffer_twomz[get_index(0)] = twoz;

        for (int m = 1; m < lmax + 1; m++) {
            int m_in_idx = get_index(m - 1);
            int m_out_idx = get_index(m);

            scalar_t c = buffer_c[m_in_idx];
            scalar_t s = buffer_s[m_in_idx];
            scalar_t twomz = buffer_twomz[m_in_idx];

            buffer_c[m_out_idx] = c * x - s * y;
            buffer_s[m_out_idx] = c * y + s * x;
            buffer_twomz[m_out_idx] = twomz + twoz;
        }
    }

    __syncthreads();

    // work through hardcoded parts first...
    int ml = min(static_cast<int>(HARDCODED_LMAX), lmax);

    clear_buffers((ml + 1) * (ml + 1), buffer_sph, buffer_dsph_x, buffer_dsph_y,
                  buffer_dsph_z, buffer_dsph_dxdx, buffer_dsph_dxdy,
                  buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
                  buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy,
                  buffer_dsph_dzdz, requires_grad, requires_hessian);

    if (threadIdx.x == 0) {
        if (lmax >= 1) {
            HARDCODED_SPH_MACRO(1, x, y, z, x2, y2, z2, buffer_sph, get_index);
            if (requires_grad) {
                HARDCODED_SPH_DERIVATIVE_MACRO(
                    1, x, y, z, x2, y2, z2, buffer_sph, buffer_dsph_x,
                    buffer_dsph_y, buffer_dsph_z, get_index);
            }

            if (requires_hessian) {
                HARDCODED_SPH_SECOND_DERIVATIVE_MACRO(
                    1, buffer_sph, buffer_dsph_dxdx, buffer_dsph_dxdy,
                    buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
                    buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy,
                    buffer_dsph_dzdz, get_index);
            }
        } else {
            COMPUTE_SPH_L0(buffer_sph, get_index);
            if (requires_grad) {
                COMPUTE_SPH_DERIVATIVE_L0(buffer_sph, buffer_dsph_x,
                                          buffer_dsph_y, buffer_dsph_z,
                                          get_index);

                if (requires_hessian) {
                    COMPUTE_SPH_SECOND_DERIVATIVE_L0(
                        buffer_sph, buffer_dsph_dxdx, buffer_dsph_dxdy,
                        buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
                        buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy,
                        buffer_dsph_dzdz, get_index);
                }
            }
        }
    }
    __syncthreads();

    // write out the values of the hardcoded derivatives from shared memory into
    // global memory.
    write_buffers(
        edge_idx, nedges, x, y, z, ir, (ml + 1) * (ml + 1), 0, buffer_sph,
        buffer_dsph_x, buffer_dsph_y, buffer_dsph_z, buffer_dsph_dxdx,
        buffer_dsph_dxdy, buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
        buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy, buffer_dsph_dzdz,
        sph, dsph, ddsph, ntotal, requires_grad, requires_hessian, normalize);

    // now lets do the generic terms for l > HARDCODED_LMAX
    int size_q = (lmax + 1) * (lmax + 2) / 2;
    int k = (HARDCODED_LMAX + 1) * (HARDCODED_LMAX + 2) / 2;
    scalar_t *qlmk = buffer_prefactors + size_q + k;
    scalar_t *pk = buffer_prefactors + k;
    int base_index = (HARDCODED_LMAX + 1) * (HARDCODED_LMAX + 1);

    for (int l = HARDCODED_LMAX + 1; l < lmax + 1; l += 1) {
        int sph_offset = l * blockDim.y;
        /*
            sph_offset needs to point to Y[l, 0], so the mapping from array
           indices to memory locations may look like: sph 0: 0, sph_offset: 0
           sph 1: 0 1 2, sph_offset: 1 sph 2: 0 1 2 3 4, sph_offset: 2 sph 3: 0
           1 2 3 4 5 6, sph_offset: 3 we also need to make sure we select the
           right atom in the buffer, hence multiplication by blockDim.y.
        */

        // clear out temporary storage buffers
        clear_buffers(2 * l + 1, buffer_sph, buffer_dsph_x, buffer_dsph_y,
                      buffer_dsph_z, buffer_dsph_dxdx, buffer_dsph_dxdy,
                      buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
                      buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy,
                      buffer_dsph_dzdz, requires_grad, requires_hessian);

        // Currently only one warp computes the spherical harmonics.
        if (threadIdx.x == 0) {
            if (requires_grad && requires_hessian) {
                generic_sph_l_channel<scalar_t, true, true, HARDCODED_LMAX,
                                      get_index>(
                    l, x, y, z, rxy, pk, qlmk, buffer_c, buffer_s, buffer_twomz,
                    buffer_sph + sph_offset, buffer_dsph_x + sph_offset,
                    buffer_dsph_y + sph_offset, buffer_dsph_z + sph_offset,
                    buffer_dsph_dxdx + sph_offset,
                    buffer_dsph_dxdy + sph_offset,
                    buffer_dsph_dxdz + sph_offset,
                    buffer_dsph_dydx + sph_offset,
                    buffer_dsph_dydy + sph_offset,
                    buffer_dsph_dydz + sph_offset,
                    buffer_dsph_dzdx + sph_offset,
                    buffer_dsph_dzdy + sph_offset,
                    buffer_dsph_dzdz + sph_offset);
            } else if (requires_grad) {
                generic_sph_l_channel<scalar_t, true, false, HARDCODED_LMAX,
                                      get_index>(
                    l, x, y, z, rxy, pk, qlmk, buffer_c, buffer_s, buffer_twomz,
                    buffer_sph + sph_offset, buffer_dsph_x + sph_offset,
                    buffer_dsph_y + sph_offset, buffer_dsph_z + sph_offset,
                    buffer_dsph_dxdx, buffer_dsph_dxdy, buffer_dsph_dxdz,
                    buffer_dsph_dydx, buffer_dsph_dydy, buffer_dsph_dydz,
                    buffer_dsph_dzdx, buffer_dsph_dzdy,
                    buffer_dsph_dzdz // these are nullpointers
                );
            } else {
                generic_sph_l_channel<scalar_t, false, false, HARDCODED_LMAX,
                                      get_index>(
                    l, x, y, z, rxy, pk, qlmk, buffer_c, buffer_s, buffer_twomz,
                    buffer_sph + sph_offset, buffer_dsph_x, buffer_dsph_y,
                    buffer_dsph_z, buffer_dsph_dxdx, buffer_dsph_dxdy,
                    buffer_dsph_dxdz, buffer_dsph_dydx, buffer_dsph_dydy,
                    buffer_dsph_dydz, buffer_dsph_dzdx, buffer_dsph_dzdy,
                    buffer_dsph_dzdz // these are nullpointers
                );
            }
        }

        // write out temporary storage buffers
        write_buffers(edge_idx, nedges, x, y, z, ir, 2 * l + 1, base_index,
                      buffer_sph, buffer_dsph_x, buffer_dsph_y, buffer_dsph_z,
                      buffer_dsph_dxdx, buffer_dsph_dxdy, buffer_dsph_dxdz,
                      buffer_dsph_dydx, buffer_dsph_dydy, buffer_dsph_dydz,
                      buffer_dsph_dzdx, buffer_dsph_dzdy, buffer_dsph_dzdz, sph,
                      dsph, ddsph, ntotal, requires_grad, requires_hessian,
                      normalize);

        base_index += 2 * l + 1;
        qlmk += l + 1;
        pk += l + 1;
    }
}

/*
    Computes the total amount of shared memory space required by
   spherical_harmonics_kernel.

    For lmax <= HARCODED_LMAX, we need to store all (HARDCODED_LMAX + 1)**2
   scalars in shared memory. For lmax > HARDCODED_LMAX, we only need to store
   each spherical harmonics vector per sample in shared memory.
*/
static size_t total_buffer_size(size_t l_max, size_t GRID_DIM_X,
                                size_t GRID_DIM_Y, size_t dtype_size,
                                bool requires_grad, bool requires_hessian) {
    int nl =
        max(static_cast<size_t>((HARDCODED_LMAX + 1) * (HARDCODED_LMAX + 1)),
            2 * l_max + 1);

    size_t total_buff_size = 0;

    total_buff_size += GRID_DIM_Y * (l_max + 1) * dtype_size; // buffer_c
    total_buff_size += GRID_DIM_Y * (l_max + 1) * dtype_size; // buffer_s
    total_buff_size += GRID_DIM_Y * (l_max + 1) * dtype_size; // buffer_twomz
    total_buff_size +=
        (l_max + 1) * (l_max + 2) * dtype_size;      // buffer_prefactors
    total_buff_size += GRID_DIM_Y * nl * dtype_size; // buffer_sph_out

    if (requires_grad) {
        total_buff_size +=
            3 * GRID_DIM_Y * nl * dtype_size; // buffer_sph_derivs
    }

    if (requires_hessian) {
        total_buff_size +=
            9 * GRID_DIM_Y * nl * dtype_size; // buffer_sph_hessian
    }

    return total_buff_size;
}

/*
    The default shared memory space on most recent NVIDIA cards is defaulted
   49152 bytes, regarldess if there is more available per SM. This method
   attempts to adjust the shared memory to fit the requested configuration if
   the kernel launch parameters exceeds the default 49152 bytes.
*/

int sphericart::cuda::adjust_shared_memory(size_t element_size, int64_t l_max,
                                           int64_t GRID_DIM_X,
                                           int64_t GRID_DIM_Y,
                                           bool requires_grad,
                                           bool requires_hessian,
                                           int64_t current_shared_mem_alloc) {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    auto required_buff_size =
        total_buffer_size(l_max, GRID_DIM_X, GRID_DIM_Y, element_size,
                          requires_grad, requires_hessian);

    if (required_buff_size > current_shared_mem_alloc &&
        required_buff_size > (deviceProp.sharedMemPerBlock -
                              deviceProp.reservedSharedMemPerBlock)) {

        if (required_buff_size > deviceProp.sharedMemPerBlockOptin) {
            return -1; // failure - need to adjust parameters
        }

        switch (element_size) {
        case 8:
            hipFuncSetAttribute(reinterpret_cast<const void*>(spherical_harmonics_kernel<double>),
                                 hipFuncAttributeMaxDynamicSharedMemorySize,
                                 required_buff_size);
            break;
        case 4:
            hipFuncSetAttribute(reinterpret_cast<const void*>(spherical_harmonics_kernel<float>),
                                 hipFuncAttributeMaxDynamicSharedMemorySize,
                                 required_buff_size);
            break;
        }

        return required_buff_size;

    } else {
        return (current_shared_mem_alloc >
                (deviceProp.sharedMemPerBlock -
                 deviceProp.reservedSharedMemPerBlock))
                   ? current_shared_mem_alloc
                   : (deviceProp.sharedMemPerBlock -
                      deviceProp.reservedSharedMemPerBlock);
    }
}

/*
    Wrapper to launch the CUDA kernel. Returns a vector containing the spherical
   harmonics and their gradients if required, otherwise returns the spherical
   harmonics and an empty tensor.

    GRID_DIM_X is the number of threads to launch in the x dimension. Used to
   parallelize over the sample dimension. GRID_DIM_Y is the number of threads to
   launch in the y dimension. Used only to improve memory throughput on reads
   and writes.

    Total number of threads used is GRID_DIM_X * GRID_DIM_Y.

    hip_stream should be of type (void *), therefore if you want to pass in
    a hipStream_t, first do void * stream_ptr = reinterpret_cast<void *>
   (stream);
*/

template <typename scalar_t>
void sphericart::cuda::spherical_harmonics_cuda_base(
    const scalar_t *__restrict__ xyz, const int nedges,
    const scalar_t *__restrict__ prefactors, const int nprefactors,
    const int64_t l_max, const bool normalize, const int64_t GRID_DIM_X,
    const int64_t GRID_DIM_Y, const bool gradients, const bool hessian,
    scalar_t *__restrict__ sph, scalar_t *__restrict__ dsph,
    scalar_t *__restrict__ ddsph, void *hip_stream) {

    int n_total = (l_max + 1) * (l_max + 1);

    dim3 grid_dim(GRID_DIM_X, GRID_DIM_Y);

    auto find_num_blocks = [](int x, int bdim) {
        return (x + bdim - 1) / bdim;
    };

    hipStream_t cstream = reinterpret_cast<hipStream_t>(hip_stream);

    dim3 block_dim(find_num_blocks(nedges, GRID_DIM_Y));

    size_t total_buff_size = total_buffer_size(
        l_max, GRID_DIM_X, GRID_DIM_Y, sizeof(scalar_t), gradients, hessian);

    spherical_harmonics_kernel<scalar_t>
        <<<block_dim, grid_dim, total_buff_size, cstream>>>(
            xyz, nedges, prefactors, nprefactors, l_max, n_total, gradients,
            hessian, normalize, sph, dsph, ddsph);

    CUDA_CHECK_KERNEL();

    CUDA_CHECK(hipStreamSynchronize(cstream));
}

template void sphericart::cuda::spherical_harmonics_cuda_base<float>(
    const float *__restrict__ xyz, const int nedges,
    const float *__restrict__ prefactors, const int nprefactors,
    const int64_t l_max, const bool normalize, const int64_t GRID_DIM_X,
    const int64_t GRID_DIM_Y, const bool gradients, const bool hessian,
    float *__restrict__ sph, float *__restrict__ dsph,
    float *__restrict__ ddsph, void *hip_stream);

template void sphericart::cuda::spherical_harmonics_cuda_base<double>(
    const double *__restrict__ xyz, const int nedges,
    const double *__restrict__ prefactors, const int nprefactors,
    const int64_t l_max, const bool normalize, const int64_t GRID_DIM_X,
    const int64_t GRID_DIM_Y, const bool gradients, const bool hessian,
    double *__restrict__ sph, double *__restrict__ dsph,
    double *__restrict__ ddsph, void *hip_stream);

/*
    CUDA kernel to computes the backwards pass for autograd.
*/
template <typename scalar_t>
__global__ void backward_kernel(const scalar_t *__restrict__ dsph,
                                const scalar_t *__restrict__ sph_grad,
                                size_t nedges, size_t n_total,
                                scalar_t *__restrict__ xyz_grad) {

    size_t edge_idx = blockIdx.x * blockDim.y + threadIdx.y;

    int spatial = blockIdx.y;

    scalar_t sum = 0.0;

    if (edge_idx < nedges) {
        // for (int j = threadIdx.x; j < sph_grad.size(1); j += blockDim.x) {
        for (int j = threadIdx.x; j < n_total; j += blockDim.x) {

            // sum += dsph[edge_idx][spatial][j] * sph_grad[edge_idx][j];
            sum += dsph[edge_idx * 3 * n_total + spatial * n_total + j] *
                   sph_grad[edge_idx * n_total + j];
        }
    }

    __syncthreads();

    // reduce across the sub-warp
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    }

    if (edge_idx < nedges) {
        if (threadIdx.x == 0) {
            // xyz_grad[sample_idx][spatial] = sum;
            xyz_grad[edge_idx * 3 + spatial] = sum;
        }
    }
}

template <typename scalar_t>
void sphericart::cuda::spherical_harmonics_backward_cuda_base(
    const scalar_t *__restrict__ dsph, const scalar_t *__restrict__ sph_grad,
    const int nedges, const int ntotal, scalar_t *__restrict__ xyz_grad,
    void *hip_stream) {

    dim3 grid_dim(4, 32);

    auto find_num_blocks = [](int x, int bdim) {
        return (x + bdim - 1) / bdim;
    };

    dim3 block_dim(find_num_blocks(nedges, 32), 3);

    hipStream_t cstream = reinterpret_cast<hipStream_t>(hip_stream);

    backward_kernel<scalar_t><<<block_dim, grid_dim, 0, cstream>>>(
        dsph, sph_grad, nedges, ntotal, xyz_grad);

    CUDA_CHECK_KERNEL();

    hipStreamSynchronize(cstream);
}

template void sphericart::cuda::spherical_harmonics_backward_cuda_base<float>(
    const float *__restrict__ dsph, const float *__restrict__ sph_grad,
    const int nedges, const int ntotal, float *__restrict__ xyz_grad,
    void *hip_stream);

template void sphericart::cuda::spherical_harmonics_backward_cuda_base<double>(
    const double *__restrict__ dsph, const double *__restrict__ sph_grad,
    const int nedges, const int ntotal, double *__restrict__ xyz_grad,
    void *hip_stream);
