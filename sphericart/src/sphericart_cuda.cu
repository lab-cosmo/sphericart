#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

#define _SPHERICART_INTERNAL_IMPLEMENTATION
#include "cuda_base.hpp"
#include "sphericart_cuda.hpp"

/*host macro that checks for errors in CUDA calls, and prints the file + line
 * and error string if one occurs
 */

using namespace std;
using namespace sphericart::cuda;

#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t cudaStatus = (call);                                       \
        if (cudaStatus != hipSuccess) {                                       \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__       \
                      << " - " << hipGetErrorString(cudaStatus) << std::endl; \
            hipDeviceReset();                                                 \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

template <typename T>
SphericalHarmonics<T>::SphericalHarmonics(size_t l_max, bool normalized) {
    /*
        This is the constructor of the SphericalHarmonics class. It initizlizes
       buffer space, compute prefactors, and sets the function pointers that are
       used for the actual calls
    */

    this->l_max = (int)l_max;
    this->nprefactors = (int)(l_max + 1) * (l_max + 2);
    this->normalized = normalized;
    this->prefactors_cpu = new T[this->nprefactors];

    // compute prefactors on host first
    compute_sph_prefactors<T>((int)l_max, this->prefactors_cpu);
    // allocate them on device and copy to device
    CUDA_CHECK(hipMalloc((void **)&this->prefactors_cuda,
                          this->nprefactors * sizeof(T)));

    CUDA_CHECK(hipMemcpy(this->prefactors_cuda, this->prefactors_cpu,
                          this->nprefactors * sizeof(T),
                          hipMemcpyHostToDevice));

    // initialise the currently available amount of shared memory.
    this->_current_shared_mem_allocation = adjust_shared_memory(
        sizeof(T), this->l_max, this->CUDA_GRID_DIM_X_, this->CUDA_GRID_DIM_Y_,
        false, false, this->_current_shared_mem_allocation);
}

template <typename T> SphericalHarmonics<T>::~SphericalHarmonics() {
    // Destructor, frees the prefactors
    delete[] (this->prefactors_cpu);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(this->prefactors_cuda));
}

template <typename T>
void SphericalHarmonics<T>::compute(const T *xyz, const size_t nsamples,
                                    bool compute_with_gradients,
                                    bool compute_with_hessian, T *sph, T *dsph,
                                    T *ddsph, void *hip_stream) {

    if (sph == nullptr) {
        throw std::runtime_error(
            "sphericart::cuda::SphericalHarmonics::compute expected "
            "sph ptr initialised, instead nullptr found. Initialise "
            "sph with hipMalloc.");
    }

    if (compute_with_gradients && dsph == nullptr) {
        throw std::runtime_error(
            "sphericart::cuda::SphericalHarmonics::compute expected "
            "dsph != nullptr since compute_with_gradients = true. "
            "initialise dsph with hipMalloc.");
    }

    if (compute_with_hessian && ddsph == nullptr) {
        throw std::runtime_error(
            "sphericart::cuda::SphericalHarmonics::compute expected "
            "ddsph != nullptr since compute_with_hessian = true. "
            "initialise ddsph with hipMalloc.");
    }

    if (this->cached_compute_with_gradients != compute_with_gradients ||
        this->cached_compute_with_hessian != compute_with_hessian) {

        this->_current_shared_mem_allocation = adjust_shared_memory(
            sizeof(T), this->l_max, this->CUDA_GRID_DIM_X_,
            this->CUDA_GRID_DIM_Y_, compute_with_gradients,
            compute_with_hessian, this->_current_shared_mem_allocation);

        if (this->_current_shared_mem_allocation == -1) {

            std::cerr << "Warning: Failed to update shared memory size, "
                         "re-attempting with  GRID_DIM_Y = 4\n"
                      << std::endl;

            this->CUDA_GRID_DIM_Y_ = 4;
            this->_current_shared_mem_allocation = adjust_shared_memory(
                sizeof(T), this->l_max, this->CUDA_GRID_DIM_X_,
                this->CUDA_GRID_DIM_Y_, compute_with_gradients,
                compute_with_hessian, this->_current_shared_mem_allocation);

            if (this->_current_shared_mem_allocation == -1) {
                throw std::runtime_error(
                    "Insufficient shared memory available to compute "
                    "spherical_harmonics with requested parameters.");
            }
        }

        this->cached_compute_with_gradients = compute_with_gradients;
        this->cached_compute_with_hessian = compute_with_hessian;
    }

    sphericart::cuda::spherical_harmonics_cuda_base<T>(
        xyz, nsamples, this->prefactors_cuda, this->nprefactors, this->l_max,
        this->normalized, this->CUDA_GRID_DIM_X_, this->CUDA_GRID_DIM_Y_,
        compute_with_gradients, compute_with_hessian, sph, dsph, ddsph,
        hip_stream);
}

// instantiates the SphericalHarmonics class for basic floating point types
template class sphericart::cuda::SphericalHarmonics<float>;
template class sphericart::cuda::SphericalHarmonics<double>;
